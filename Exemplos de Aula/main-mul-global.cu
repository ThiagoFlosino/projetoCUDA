
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

#define BLOCK_SIZE 32

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                     \
      fprintf(stderr,"CUDA ERROR:%s in file: %s in line: ", hipGetErrorString(call),  __FILE__, __LINE__); \
         exit(0);                                                                                 \
   } } while (0)


// Set a matrix element
__device__ __forceinline__  int getPosition(int row, int col)
{
	return row * (gridDim.x * blockDim.x) + col;
}


/*
 * Multiplicação usando memória global
 */

__global__ void multMatrixG(float *C, float *A, float *B)
{
    float Cvalue = 0.0f;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //int width =  (gridDim.x * blockDim.x);
    //int height = (gridDim.y * blockDim.y);


    for (int e = 0; e < (gridDim.x * blockDim.x); ++e){
      Cvalue += A[getPosition(row, e)] * B[getPosition(e, col)];
    }


    C[getPosition(row, col)] = Cvalue;// (float) height;
}



void printMatrix(float *m, float w, float h){
   int i, j;

   printf("\n");

   for (j = 0; j < h; j++){
      for (i = 0; i < w; i++){
         int k = j * w + i;
         printf("%.2f ", m[k]);
      }
      printf("\n");
   }

}

int main (int argc, char **argv){

   float *h_A, *h_B, *h_C;
   int iC, jC;

   int width      = atoi(argv[1]);
   int height     = width;
   int GPU        = 0;

   float   *d_C = NULL,
           *d_A = NULL,
           *d_B = NULL;

   hipEvent_t start; 
   hipEvent_t stop;  
   
   float GPUTime = 0.0f,
         MEMTime = 0.0f,
         aux     = 0.0f;

   srand (time(NULL));

   printf("\nMultiplicando matriz - GPU\n");
   printf("Tamanho da matriz: %d x %d \n", width, height);

    h_A = (float*) malloc (width * height * sizeof(float));
    h_B = (float*) malloc (width * height * sizeof(float));
    h_C = (float*) malloc (width * height * sizeof(float));


   for (jC = 0; jC < height; jC++){
      for (iC = 0; iC < width; iC++){
         int kC = jC * width + iC;
         h_A[kC] = (float) (rand() % 65536 + 1) / 65536.0f;

         if (jC == iC)
           h_B[kC] = 1.0f;
         else
            h_B[kC] = 0.0f;

      }
   }

	CHECK_ERROR(hipSetDevice(GPU));

	//Reset na GPU selecionada
	CHECK_ERROR(hipDeviceReset());

   CHECK_ERROR(hipEventCreate(&start));
   CHECK_ERROR(hipEventCreate(&stop));
   

   //Aloca memória GPU
   CHECK_ERROR(hipMalloc((void**) &d_A, width * height * sizeof(float)));
   CHECK_ERROR(hipMalloc((void**) &d_B, width * height * sizeof(float)));
   CHECK_ERROR(hipMalloc((void**) &d_C, width * height * sizeof(float)));

   //Copiando CPU --> GPU
   CHECK_ERROR(hipEventRecord(start, 0));
   
   CHECK_ERROR(hipMemcpy(d_A, h_A, width * height * sizeof(float),  hipMemcpyHostToDevice));
   CHECK_ERROR(hipMemcpy(d_B, h_B, width * height * sizeof(float),  hipMemcpyHostToDevice));

   CHECK_ERROR(hipEventRecord(stop, 0));
   CHECK_ERROR(hipEventSynchronize(stop));
   CHECK_ERROR(hipEventElapsedTime(&aux, start, stop));

   MEMTime = aux;
      

   
      //int numBlocks = 1;
   //int threadsPerBlock = WIDTH*HEIGHT / numBlocks;

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y, 1);


   CHECK_ERROR(hipEventRecord(start, 0));
	multMatrixG <<<numBlocks, threadsPerBlock >>> (d_C, d_A, d_B);
	CHECK_ERROR(hipDeviceSynchronize());
   CHECK_ERROR(hipEventRecord(stop, 0));
   CHECK_ERROR(hipEventSynchronize(stop));
   CHECK_ERROR(hipEventElapsedTime(&GPUTime, start, stop));

      
      
   CHECK_ERROR(hipEventRecord(start, 0));
	CHECK_ERROR(hipMemcpy(h_C, d_C,  width*height * sizeof(float),  hipMemcpyDeviceToHost));
   CHECK_ERROR(hipEventRecord(stop, 0));
   CHECK_ERROR(hipEventSynchronize(stop));
   CHECK_ERROR(hipEventElapsedTime(&aux, start, stop));

   MEMTime += aux;
	
	fprintf(stderr, "\n %f %f %f", MEMTime, GPUTime, (MEMTime + GPUTime));
	fprintf(stdout, "\n MEM:\t %f \nGPU:\t %f \n MEM+GPU:\t %f", MEMTime, GPUTime, (MEMTime + GPUTime));
	

   CHECK_ERROR(hipEventDestroy(start));
   CHECK_ERROR(hipEventDestroy(stop));
   CHECK_ERROR(hipFree(d_A));  //Liberando memorias GPU e CPU
   CHECK_ERROR(hipFree(d_B));  //Liberando memorias GPU e CPU
   CHECK_ERROR(hipFree(d_C));  //Liberando memorias GPU e CPU

	
   float err = 0.0f;
   for (jC = 0; jC < height; jC++){
        for (iC = 0; iC < width; iC++){
           int kC = jC * width + iC;
           if (fabs(h_A[kC]-h_C[kC]) > 0.000000001f)
        	   err++;
        }
   }
   fprintf(stdout, "\nError: %f\n", (err / (float)(width*height)));
   //validando

   //printMatrix(h_A, width, height);
   //printMatrix(h_C, width, height);

   free(h_A);
   free(h_B);
   free(h_C);

   fprintf(stdout, "FIM\n");

   return EXIT_SUCCESS;
}
